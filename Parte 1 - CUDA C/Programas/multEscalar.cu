
#include <hip/hip_runtime.h>
#include "stdio.h"
#define N 100

__global__ void add(int *a, int *c)
{
  int tID   = blockIdx.x;
  if (tID < N)
  {
	c[tID] = 3*a[tID];
  }
}

int main()
{
	
  int a[N], c[N];
  int *d_a, *d_c;

  hipMalloc((void **) &d_a, N*sizeof(int));
  hipMalloc((void **) &d_c, N*sizeof(int));

  // Llenar el arreglo
  for (int i = 0; i < N; i++)
  {
	a[i] = i;
  }

  hipMemcpy(d_a, a, N*sizeof(int), hipMemcpyHostToDevice);

  add<<<N,1>>>(d_a, d_c);

  hipMemcpy(c, d_c, N*sizeof(int), hipMemcpyDeviceToHost);

  for (int i = 0; i < N; i++)
  {
	printf("3*%d = %d\n", a[i], c[i]);
  }
  
  return 0;
  
}
